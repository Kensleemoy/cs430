#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>

typedef unsigned char BYTE;

#define IMAGE_SIZE 6*1000*1000
#define MAXITER 1000
#define X_RES 1000
#define Y_RES 1000

/**
 * writeOutput
 * 
 * Write Mandelbrot image in PGM format
 * 
 * @param fileName - Filename to write PGM image
 * @param image - output array image (Mandelbrot pixels)
 * @param width - image width
 * @param height - image height
 * 
 * */
 void writeOutput(const char *fileName, BYTE *image, int width, int height) {
   double xmin = -2;
   double xmax = 1;
   double ymin = -1.5;
   double ymax = 1.5;

   FILE *fp = fopen(fileName, "wb");
    fprintf(fp,"P6\n# Mandelbrot, xmin=%lf, xmax=%lf, ymin=%lf, ymax=%lf, maxiter=%d\n%d\n%d\n%d\n", xmin, xmax, ymin, ymax, MAXITER, X_RES, Y_RES, (MAXITER < 256 ? 256 : MAXITER));
    fwrite(image, 1, IMAGE_SIZE, fp);
    fclose(fp);
}

__global__ void mandelbrot(BYTE* image,uint16_t maxiter){
    	double xmin = -2;
    	double xmax = 1;
    	double ymin = -1.5;
    	double ymax = 1.5;
    	double dx = (xmax-xmin)/X_RES;
    	double dy = (ymax-ymin)/Y_RES;

	//int index_x = blockIdx.x * blockDim.x + threadIdx.x;
	//int index_y = blockIdx.y * blockDim.y + threadIdx.y;
	//int grid_width = gridDim.x * blockDim.x;
	//int index = index_x * grid_width + index_y;

	//int j = blockIdx.x * blockIdx.y;
	int j = blockIdx.y * gridDim.x + blockIdx.x;
	//int i = threadIdx.x * threadIdx.y;
	int i = threadIdx.y * blockDim.x + threadIdx.x;

	double x = xmin + i * dx;
	double y = ymax - j * dy;

	int k;
	double u = 0.0;
	double v = 0.0;
	double u2 = u*u;
	double v2 = v*v;

        for (k = 1; k < maxiter && (u2 + v2 < 4.0); k++) {
            v = 2 * u * v + y;
            u = u2 - v2 + x;
            u2 = u * u;
            v2 = v * v;
	}
	
	int pxlStartLoc = 6*((j*X_RES)+i);
	//int pxlStartLoc = 6*(index);
	if (k >= maxiter) {
		image[pxlStartLoc+0] = 0;
        	image[pxlStartLoc+1] = 0;
        	image[pxlStartLoc+2] = 0;
        	image[pxlStartLoc+3] = 0;
        	image[pxlStartLoc+4] = 0;
        	image[pxlStartLoc+5] = 0;
      	}
	else {
        	image[pxlStartLoc+0] = k >> 8;
        	image[pxlStartLoc+1] = k & 255;
        	image[pxlStartLoc+2] = k >> 8;
        	image[pxlStartLoc+3] = k & 255;
        	image[pxlStartLoc+4] = k >> 8;
        	image[pxlStartLoc+5] = k & 255;
	}	
}

int main(int argc, char* argv[]) {
	

	BYTE* image;
	dim3 grid_dim(100,10,1);
	dim3 block_dim(100,10,1);

	hipMallocManaged(&image, IMAGE_SIZE);	
	mandelbrot<<<grid_dim,block_dim>>>(image,MAXITER);
	hipDeviceSynchronize();

	writeOutput(argv[1],image,X_RES,Y_RES);
	hipFree(image);
	return 0; 
}
