#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>

typedef unsigned char BYTE;

#define IMAGE_SIZE 6*1000*1000
#define X_RES 1000
#define Y_RES 1000

/**
 * writeOutput
 * 
 * Write Mandelbrot image in PGM format
 * 
 * @param fileName - Filename to write PGM image
 * @param image - output array image (Mandelbrot pixels)
 * @param width - image width
 * @param height - image height
 * 
 * */
 void writeOutput(const char *fileName, BYTE *image, int width, int height) {
    int i,j;
    int max = -1;
    int size = width*height;

    for (i=0; i < size; ++i) {
        if(image[i] > max) {
            max = image[i];
        }
    }

    FILE *fout = fopen(fileName, "w");

    fprintf(fout, "P2\n");
    fprintf(fout, "%d\t%d\n", width, height);
    fprintf(fout, "%d\n", max);

    for (i=0; i < height; ++i) {
        for (j=0; j<width; ++j) {
            fprintf(fout, "%d\t", image[i*width+j]);
        }
        fprintf(fout,"\n");
    }

    fflush(fout);
    fclose(fout);
}

__global__ void mandelbrot(BYTE* image,uint16_t maxiter,double u,double v,double x,double y){
	int k;
	double u2 = u*u;
	double v2 = v*v;
    	double xmin = -2;
    	double xmax = 1;
    	double ymin = -1.5;
    	double ymax = 1.5;
    	double dx = (xmax-xmin)/X_RES;
    	double dy = (ymax-ymin)/Y_RES;

        for (k = 1; k < maxiter && (u2 + v2 < 4.0); k++) {
            v = 2 * u * v + y;
            u = u2 - v2 + x;
            u2 = u * u;
            v2 = v * v;
	}

	int pxlStartLoc = 6*((j*xres)+i);//TODO Calculate index for CUDA
	if (k >= maxiter) {
		image[pxlStartLoc+0] = 0;
        	image[pxlStartLoc+1] = 0;
        	image[pxlStartLoc+2] = 0;
        	image[pxlStartLoc+3] = 0;
        	image[pxlStartLoc+4] = 0;
        	image[pxlStartLoc+5] = 0;
      	}
	else {
        	image[pxlStartLoc+0] = k >> 8;
        	image[pxlStartLoc+1] = k & 255;
        	image[pxlStartLoc+2] = k >> 8;
        	image[pxlStartLoc+3] = k & 255;
        	image[pxlStartLoc+4] = k >> 8;
        	image[pxlStartLoc+5] = k & 255;
	}	
}

int main(int argc, char* argv[]) {
	

	BYTE* image;
	dim3 grid_dim(100,1,1);//TODO gridDim
	dim3 block_dim(10,1,1);//TODO blockDim

	hipMalloc(&image, IMAGE_SIZE);	
	mandelbrot<<<grid_dim,block_dim>>>(image,1000,?,?,?,?);//TODO add dimensions
	hipDeviceSynchronize();

	writeOutput(argv[1],image,X_RES,Y_RES);
	hipFree(image)
	return 0; 
}
